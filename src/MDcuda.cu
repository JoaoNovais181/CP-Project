/*
 MD.c - a simple molecular dynamics program for simulating real gas properties of Lennard-Jones particles.
 
 Copyright (C) 2016  Jonathan J. Foley IV, Chelsea Sweet, Oyewumi Akinfenwa
 
 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program.  If not, see <http://www.gnu.org/licenses/>.
 
 Electronic Contact:  foleyj10@wpunj.edu
 Mail Contact:   Prof. Jonathan Foley
 Department of Chemistry, William Paterson University
 300 Pompton Road
 Wayne NJ 07470
 
 */
#include <cuda_device_runtime_api.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>

int THREADS_PER_BLOCK = 64;

// Number of particles
int N;

//  Lennard-Jones parameters in natural units!
double sigma = 1.;
double epsilon = 1.;
double m = 1.;
double kB = 1.;

double NA = 6.022140857e23;
double kBSI = 1.38064852e-23;  // m^2*kg/(s^2*K)

//  Size of box, which will be specified in natural units
double L;

//  Initial Temperature in Natural Units
double Tinit;  //2;
//  Vectors!
//
const int MAXPART=10001;

typedef struct vect3d
{
    double x, y, z;
} Vect3d;

//  Position
Vect3d r[MAXPART];
//  Velocity
Vect3d v[MAXPART];
//  Acceleration
Vect3d a[MAXPART];
//  Force
Vect3d F[MAXPART];

// #if __CUDA_ARCH__ < 600
__device__ 
double atomicAdd2(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
// #endif

// atom type
char atype[10];
//  Function prototypes
//  initialize positions on simple cubic lattice, also calls function to initialize velocities
void initialize();  
//  update positions and velocities using Velocity Verlet algorithm 
//  print particle coordinates to file for rendering via VMD or other animation software
//  return 'instantaneous pressure'
double VelocityVerlet(double dt, int iter, FILE *fp, double *POT);  
//  Compute Force using F = -dV/dr
//  solve F = ma for use in Velocity Verlet
void computeAccelerations();
//  Compute Force using F = -dV/dr
//  solve F = ma for use in Velocity Verlet
//  Compute total potential energy from particle coordinates
double computeAccelerationsAndPotential();
//  Numerical Recipes function for generation gaussian distribution
double gaussdist();
//  Initialize velocities according to user-supplied initial Temperature (Tinit)
void initializeVelocities();
//  Compute mean squared velocity from particle velocities
double MeanSquaredVelocity();
//  Compute total kinetic energy from particle mass and velocities
double Kinetic();

__host__ __device__
double myPow(double base, int exp);

int main(int argc, char *argv[])
{

    
    //  variable delcarations
    int i;
    double dt, Vol, Temp, Press, Pavg, Tavg, rho;
    double VolFac, TempFac, PressFac, timefac;
    double KE, PE, mvs, gc, Z;
    char trash[10000], prefix[1000], tfn[1000], ofn[1000], afn[1000];
    FILE *infp, *tfp, *ofp, *afp;
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  WELCOME TO WILLY P CHEM MD!\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  ENTER A TITLE FOR YOUR CALCULATION!\n");
    scanf("%s",prefix);
    strcpy(tfn,prefix);
    strcat(tfn,"_traj.xyz");
    strcpy(ofn,prefix);
    strcat(ofn,"_output.txt");
    strcpy(afn,prefix);
    strcat(afn,"_average.txt");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  TITLE ENTERED AS '%s'\n",prefix);
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    /*     Table of values for Argon relating natural units to SI units:
     *     These are derived from Lennard-Jones parameters from the article
     *     "Liquid argon: Monte carlo and molecular dynamics calculations"
     *     J.A. Barker , R.A. Fisher & R.O. Watts
     *     Mol. Phys., Vol. 21, 657-673 (1971)
     *
     *     mass:     6.633e-26 kg          = one natural unit of mass for argon, by definition
     *     energy:   1.96183e-21 J      = one natural unit of energy for argon, directly from L-J parameters
     *     length:   3.3605e-10  m         = one natural unit of length for argon, directly from L-J parameters
     *     volume:   3.79499-29 m^3        = one natural unit of volume for argon, by length^3
     *     time:     1.951e-12 s           = one natural unit of time for argon, by length*sqrt(mass/energy)
     ***************************************************************************************/
    
    //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //  Edit these factors to be computed in terms of basic properties in natural units of
    //  the gas being simulated
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("  WHICH NOBLE GAS WOULD YOU LIKE TO SIMULATE? (DEFAULT IS ARGON)\n");
    printf("\n  FOR HELIUM,  TYPE 'He' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR NEON,    TYPE 'Ne' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR ARGON,   TYPE 'Ar' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR KRYPTON, TYPE 'Kr' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR XENON,   TYPE 'Xe' THEN PRESS 'return' TO CONTINUE\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    scanf("%s",atype);
    
    if (strcmp(atype,"He")==0) {
        
        VolFac = 1.8399744000000005e-29;
        PressFac = 8152287.336171632;
        TempFac = 10.864459551225972;
        timefac = 1.7572698825166272e-12;
        
    }
    else if (strcmp(atype,"Ne")==0) {
        
        VolFac = 2.0570823999999997e-29;
        PressFac = 27223022.27659913;
        TempFac = 40.560648991243625;
        timefac = 2.1192341945685407e-12;
        
    }
    else if (strcmp(atype,"Ar")==0) {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        //strcpy(atype,"Ar");
        
    }
    else if (strcmp(atype,"Kr")==0) {
        
        VolFac = 4.5882712000000004e-29;
        PressFac = 59935428.40275003;
        TempFac = 199.1817584391428;
        timefac = 8.051563913585078e-13;
        
    }
    else if (strcmp(atype,"Xe")==0) {
        
        VolFac = 5.4872e-29;
        PressFac = 70527773.72794868;
        TempFac = 280.30305642163006;
        timefac = 9.018957925790732e-13;
        
    }
    else {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        strcpy(atype,"Ar");
        
    }
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n                     YOU ARE SIMULATING %s GAS! \n",atype);
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  YOU WILL NOW ENTER A FEW SIMULATION PARAMETERS\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n\n  ENTER THE INTIAL TEMPERATURE OF YOUR GAS IN KELVIN\n");
    scanf("%lf",&Tinit);
    // Make sure temperature is a positive number!
    if (Tinit<0.) {
        printf("\n  !!!!! ABSOLUTE TEMPERATURE MUST BE A POSITIVE NUMBER!  PLEASE TRY AGAIN WITH A POSITIVE TEMPERATURE!!!\n");
        exit(0);
    }
    // Convert initial temperature from kelvin to natural units
    Tinit /= TempFac;
    
    
    printf("\n\n  ENTER THE NUMBER DENSITY IN moles/m^3\n");
    printf("  FOR REFERENCE, NUMBER DENSITY OF AN IDEAL GAS AT STP IS ABOUT 40 moles/m^3\n");
    printf("  NUMBER DENSITY OF LIQUID ARGON AT 1 ATM AND 87 K IS ABOUT 35000 moles/m^3\n");
    
    scanf("%lf",&rho);
    
    // N = 10*216;
    N = 5000;
    if (argc >= 3)
    {
        N = atoi(argv[2]);
    }
    if (argc >= 2)
    {
        THREADS_PER_BLOCK = atoi(argv[1]);
    }

    Vol = N/(rho*NA);
    
    Vol /= VolFac;
    
    //  Limiting N to MAXPART for practical reasons
    if (N>=MAXPART) {
        
        printf("\n\n\n  MAXIMUM NUMBER OF PARTICLES IS %i\n\n  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY \n\n", MAXPART);
        exit(0);
        
    }
    //  Check to see if the volume makes sense - is it too small?
    //  Remember VDW radius of the particles is 1 natural unit of length
    //  and volume = L*L*L, so if V = N*L*L*L = N, then all the particles
    //  will be initialized with an interparticle separation equal to 2xVDW radius
    if (Vol<N) {
        
        printf("\n\n\n  YOUR DENSITY IS VERY HIGH!\n\n");
        printf("  THE NUMBER OF PARTICLES IS %i AND THE AVAILABLE VOLUME IS %f NATURAL UNITS\n",N,Vol);
        printf("  SIMULATIONS WITH DENSITY GREATER THAN 1 PARTCICLE/(1 Natural Unit of Volume) MAY DIVERGE\n");
        printf("  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY AND RETRY\n\n");
        exit(0);
    }
    // Vol = L*L*L;
    // Length of the box in natural units:
    L = pow(Vol,(1./3));
    
    //  Files that we can write different quantities to
    tfp = fopen(tfn,"w");     //  The MD trajectory, coordinates of every particle at each timestep
    ofp = fopen(ofn,"w");     //  Output of other quantities (T, P, gc, etc) at every timestep
    afp = fopen(afn,"w");    //  Average T, P, gc, etc from the simulation
    
    int NumTime;
    if (strcmp(atype,"He")==0) {
        
        // dt in natural units of time s.t. in SI it is 5 f.s. for all other gasses
        dt = 0.2e-14/timefac;
        //  We will run the simulation for NumTime timesteps.
        //  The total time will be NumTime*dt in natural units
        //  And NumTime*dt multiplied by the appropriate conversion factor for time in seconds
        NumTime=50000;
    }
    else {
        dt = 0.5e-14/timefac;
        NumTime=200;
        
    }
    
    //  Put all the atoms in simple crystal lattice and give them random velocities
    //  that corresponds to the initial temperature we have specified
    initialize();
    
    //  Based on their positions, calculate the ininial intermolecular forces
    //  The accellerations of each particle will be defined from the forces and their
    //  mass, and this will allow us to update their positions via Newton's law
    computeAccelerations();
    
    
    // Print number of particles to the trajectory file
    fprintf(tfp,"%i\n",N);
    
    //  We want to calculate the average Temperature and Pressure for the simulation
    //  The variables need to be set to zero initially
    Pavg = 0;
    Tavg = 0;
    
    
    int tenp = floor(NumTime/10);
    fprintf(ofp,"  time (s)              T(t) (K)              P(t) (Pa)           Kinetic En. (n.u.)     Potential En. (n.u.) Total En. (n.u.)\n");
    printf("  PERCENTAGE OF CALCULATION COMPLETE:\n  [");
    for (i=0; i<NumTime+1; i++) {
        
        //  This just prints updates on progress of the calculation for the users convenience
        if (i==tenp) printf(" 10 |");
        else if (i==2*tenp) printf(" 20 |");
        else if (i==3*tenp) printf(" 30 |");
        else if (i==4*tenp) printf(" 40 |");
        else if (i==5*tenp) printf(" 50 |");
        else if (i==6*tenp) printf(" 60 |");
        else if (i==7*tenp) printf(" 70 |");
        else if (i==8*tenp) printf(" 80 |");
        else if (i==9*tenp) printf(" 90 |");
        else if (i==10*tenp) printf(" 100 ]\n");
        fflush(stdout);
        
        
        // This updates the positions and velocities using Newton's Laws
        // Also computes the Pressure as the sum of momentum changes from wall collisions / timestep
        // which is a Kinetic Theory of gasses concept of Pressure
        Press = VelocityVerlet(dt, i+1, tfp, &PE);
        Press *= PressFac;
        
        //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        //  Now we would like to calculate somethings about the system:
        //  Instantaneous mean velocity squared, Temperature, Pressure
        //  Potential, and Kinetic Energy
        //  We would also like to use the IGL to try to see if we can extract the gas constant
        mvs = MeanSquaredVelocity();
        KE = Kinetic();
        
        // Temperature from Kinetic Theory
        Temp = m*mvs/(3*kB) * TempFac;
        
        // Instantaneous gas constant and compressibility - not well defined because
        // pressure may be zero in some instances because there will be zero wall collisions,
        // pressure may be very high in some instances because there will be a number of collisions
        gc = NA*Press*(Vol*VolFac)/(N*Temp);
        Z  = Press*(Vol*VolFac)/(N*kBSI*Temp);
        
        Tavg += Temp;
        Pavg += Press;

        fprintf(ofp,"  %8.4e  %20.8f  %20.8f %20.8f  %20.8f  %20.8f \n",i*dt*timefac,Temp,Press,KE, PE, KE+PE);
        //fprintf(ofp,"  %8.4e  %20.8f  %20.4f %20.8f  %20.7f  %20.7f \n",i*dt*timefac,Temp,Press,KE, PE, KE+PE);
        
        
    }
    
    // Because we have calculated the instantaneous temperature and pressure,
    // we can take the average over the whole simulation here
    Pavg /= NumTime;
    Tavg /= NumTime;
    Z = Pavg*(Vol*VolFac)/(N*kBSI*Tavg);
    gc = NA*Pavg*(Vol*VolFac)/(N*Tavg);
    fprintf(afp,"  Total Time (s)      T (K)               P (Pa)      PV/nT (J/(mol K))         Z           V (m^3)              N\n");
    fprintf(afp," --------------   -----------        ---------------   --------------   ---------------   ------------   -----------\n");
    fprintf(afp,"  %8.4e  %15.5f       %15.5f     %10.5f       %10.5f        %10.5e         %i\n",i*dt*timefac,Tavg,Pavg,gc,Z,Vol*VolFac,N);
    
    printf("\n  TO ANIMATE YOUR SIMULATION, OPEN THE FILE \n  '%s' WITH VMD AFTER THE SIMULATION COMPLETES\n",tfn);
    printf("\n  TO ANALYZE INSTANTANEOUS DATA ABOUT YOUR MOLECULE, OPEN THE FILE \n  '%s' WITH YOUR FAVORITE TEXT EDITOR OR IMPORT THE DATA INTO EXCEL\n",ofn);
    printf("\n  THE FOLLOWING THERMODYNAMIC AVERAGES WILL BE COMPUTED AND WRITTEN TO THE FILE  \n  '%s':\n",afn);
    printf("\n  AVERAGE TEMPERATURE (K):                 %15.5f\n",Tavg);
    printf("\n  AVERAGE PRESSURE  (Pa):                  %15.5f\n",Pavg);
    printf("\n  PV/nT (J * mol^-1 K^-1):                 %15.5f\n",gc);
    printf("\n  PERCENT ERROR of pV/nT AND GAS CONSTANT: %15.5f\n",100*fabs(gc-8.3144598)/8.3144598);
    printf("\n  THE COMPRESSIBILITY (unitless):          %15.5f \n",Z);
    printf("\n  TOTAL VOLUME (m^3):                      %10.5e \n",Vol*VolFac);
    printf("\n  NUMBER OF PARTICLES (unitless):          %i \n", N);
    
    
    
    
    fclose(tfp);
    fclose(ofp);
    fclose(afp);
    
    return 0;
}


void initialize() {
    int n, p, i, j, k;
    double pos;
    
    // Number of atoms in each direction
    n = int(ceil(pow(N, 1.0/3)));
    
    //  spacing between atoms along a given direction
    pos = L / n;
    
    //  index for number of particles assigned positions
    p = 0;
    //  initialize positions
    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            for (k=0; k<n; k++) {
                if (p<N) {
                    r[p].x = (i + 0.5)*pos;
                    r[p].y = (j + 0.5)*pos;
                    r[p].z = (k + 0.5)*pos;
                }
                p++;
            }
        }
    }
    
    // Call function to initialize velocities
    initializeVelocities();
    
    /***********************************************
     *   Uncomment if you want to see what the initial positions and velocities are
     printf("  Printing initial positions!\n");
     for (i=0; i<N; i++) {
     printf("  %6.3e  %6.3e  %6.3e\n",r[i].x,r[i].y,r[i].z);
     }
     
     printf("  Printing initial velocities!\n");
     for (i=0; i<N; i++) {
     printf("  %6.3e  %6.3e  %6.3e\n",v[i].x,v[i].y,v[i].z);
     }
     */
    
    
    
}   


__host__ __device__
double myPow(double base, int exp)
{
    if (exp > 0)
        return base * myPow(base, exp-1);
    if (exp == 0)
        return 1;
    return 1 / myPow(base, -exp);
}

//  Function to calculate the averaged velocity squared
double MeanSquaredVelocity() { 
    
    double vx2 = 0;
    double vy2 = 0;
    double vz2 = 0;
    double v2;
    
    for (int i=0; i<N; i++) {
        Vect3d vect = v[i];
        vx2 = vx2 + vect.x*vect.x;
        vy2 = vy2 + vect.y*vect.y;
        vz2 = vz2 + vect.z*vect.z;
    }
    v2 = (vx2+vy2+vz2)/N;

    // printf("  Average of x-component of velocity squared is %f\n",v2);
    return v2;
}

//  Function to calculate the kinetic energy of the system
double Kinetic() { //Write Function here!  
    
    double v2, kin;
    
    kin =0.;
    for (int i=0; i<N; i++) {
        
        Vect3d vect = v[i];
        v2 = (vect.x*vect.x) + (vect.y*vect.y) + (vect.z*vect.z);
        kin += m*v2/2;
        
    }
    
    // printf("  Total Kinetic Energy is %f\n",N*kin*m/2.);
    return kin;
    
}

void checkCUDAError (const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s, %s", msg, hipGetErrorString( err));
        exit(-1);
    }
}

__global__
void computeAccelerationsAndPotentialKernel(Vect3d *dr, Vect3d *da, double *dPot, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int threadID = threadIdx.x;

    double Pot=0, f, x, y, z, ax, ay, az, rx, ry, rz, r2, r6, r8;
    int j;
    Vect3d riVect, rjVect;
    extern __shared__ double potential[];

    potential[threadID] = 0.0;

    if (id < N)
    {
        riVect = dr[id];
        rx = riVect.x; ry = riVect.y; rz = riVect.z;
        ax = 0; ay = 0; az = 0;
        for (j=0; j<N; j++)
        {
            if (id != j)
            {
                rjVect = dr[j];

                x = rx-rjVect.x;
                y = ry-rjVect.y;
                z = rz-rjVect.z;

                r2 = x*x + y*y + z*z;
                r8 = myPow(r2, 4);
                r6 = myPow(r2, 3);

                f = 24 * (2 - r6) / (r8*r6);

                Pot += (1-r6) / (r6*r6);

                x = x*f; y = y*f; z = z*f;

                ax += x;
                ay += y;
                az += z;
            }
        }
        da[id] = {ax, ay, az};

        potential[threadID] = 4*Pot;

        __syncthreads();

        if(threadID == 0){
            for (int i=blockDim.x-2 ; i>=0 ; i--)
                potential[i] += potential[i+1];
            atomicAdd2(dPot, potential[0]);
        }
    }
}

double launchComputeAccelerationsAndPotencialKernel()
{
    Vect3d *dr1;
    Vect3d *da1;
    double Pot1, Pot2, *dPot1, *dPot2;

    size_t sizeStruct = N * sizeof(struct vect3d),
           sizePot = 1 * sizeof(double);

    hipStream_t stream1; 
    hipStreamCreate(&stream1);
    checkCUDAError("stream creation");

    hipMallocAsync((void **) &dr1, sizeStruct, stream1);
    hipMallocAsync((void **) &da1, sizeStruct, stream1);
    hipMallocAsync((void **) &dPot1, sizePot, stream1);

    checkCUDAError("memory allocation");

    hipMemcpyAsync(dr1, r, sizeStruct, hipMemcpyHostToDevice, stream1);
    hipMemsetAsync(da1, 0, sizeStruct, stream1);
    hipMemsetAsync(dPot1, 0, sizePot, stream1);

    checkCUDAError("memcpy host->device");

    int numBlocks = N/THREADS_PER_BLOCK + (N%THREADS_PER_BLOCK != 0);
    int sharedMemorySize = THREADS_PER_BLOCK * sizeof(double);

    computeAccelerationsAndPotentialKernel <<< numBlocks, THREADS_PER_BLOCK, sharedMemorySize, stream1 >>> (dr1, da1, dPot1, N);
    checkCUDAError("kernel invocation");

    hipStreamSynchronize(stream1);

    hipMemcpyAsync(a, da1, sizeStruct, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(&Pot1, dPot1, sizePot, hipMemcpyDeviceToHost, stream1);


    checkCUDAError("memcpy device->host");

    hipDeviceSynchronize();

    hipFree(dr1);
    hipFree(da1);
    hipFree(dPot1);
    checkCUDAError("mem free");

    return Pot1;
}

//   Uses the derivative of the Lennard-Jones potential to calculate
//   the forces on each atom.  Then uses a = F/m to calculate the
//   accelleration of each atom. 
void computeAccelerations() {
    int i, j;
    double f, rSqd, x, y, z, ax, ay, az, xr, yr, zr;
    Vect3d riVect, rjVect, ai; // position of i relative to j
    
    
    for (i = 0; i < N; i++) {  // set all accelerations to zero
        a[i] = {0.0, 0.0, 0.0};
    }

    for (i = 0; i < N-1; i++) {   // loop over all distinct pairs i,j
        riVect = r[i]; 
        xr = riVect.x;
        yr = riVect.y;
        zr = riVect.z;
        ai = a[i];
        ax = ai.x;
        ay = ai.y;
        az = ai.z;
        for (j = i+1; j < N; j++) {
            // initialize r^2 to zero
            rjVect = r[j];
            x = xr-rjVect.x;
            y = yr-rjVect.y;
            z = zr-rjVect.z;

            rSqd = (x*x) + (y*y) + (z*z);

            //  From derivative of Lennard-Jones with sigma and epsilon set equal to 1 in natural units!
            double aux = myPow(rSqd, 4), aux2 = myPow(rSqd, 3);
            // f = 24 * (2 * aux*aux2 - aux);
            // f = 24 * aux * (2*aux2 - 1);
            f = 24 * ((2 - aux2) / (aux*aux2));

            x = x*f; y = y*f; z = z*f;

            ax += x;
            ay += y;
            az += z;

            a[j].x -= x;
            a[j].y -= y;
            a[j].z -= z;
        }
        a[i] = {ax, ay, az};
    }
}

double VelocityVerlet(double dt, int iter, FILE *fp, double *POT) {
    int i;
    Vect3d vel, acl;
    
    double psum = 0., dtSqd = dt*dt, halfDt = 0.5*dt;
    double ax, ay, az;
    
    //  Compute accelerations from forces at current position
    // this call was removed (commented) for predagogical reasons
    //computeAccelerations();
    //  Update positions and velocity with current velocity and acceleration
    //printf("  Updated Positions!\n");
    for (i=0; i<N; i++) {
        vel = v[i]; acl = a[i];
        ax = acl.x; ay = acl.y; az = acl.z;
        r[i].x += vel.x*dt + 0.5*ax*dtSqd;
        r[i].y += vel.y*dt + 0.5*ay*dtSqd;
        r[i].z += vel.z*dt + 0.5*az*dtSqd;

        v[i].x += ax*halfDt;
        v[i].y += ay*halfDt;
        v[i].z += az*halfDt;
    }
    //  Update accellerations from updated positions
    (*POT) = launchComputeAccelerationsAndPotencialKernel(); 
	//computeAccelerationsAndPotential();
    //  Update velocity with updated acceleration
    for (i=0; i<N; i++) {
        acl = a[i];
        v[i].x += acl.x*halfDt;
        v[i].y += acl.y*halfDt;
        v[i].z += acl.z*halfDt;
    }
    
    // Elastic walls
    for (i=0; i<N; i++) {
        if (r[i].x<0.) {
            v[i].x *=-1; //- elastic walls
            psum += 2*m*fabs(v[i].x)/dt;  // contribution to pressure from "left" walls
        }
        else if (r[i].x>=L) {
            v[i].x*=-1;  //- elastic walls
            psum += 2*m*fabs(v[i].x)/dt;  // contribution to pressure from "right" walls
        }
        if (r[i].y<0.) {
            v[i].y *=-1; //- elastic walls
            psum += 2*m*fabs(v[i].y)/dt;  // contribution to pressure from "left" walls
        }
        else if (r[i].y>=L) {
            v[i].y*=-1;  //- elastic walls
            psum += 2*m*fabs(v[i].y)/dt;  // contribution to pressure from "right" walls
        }
        if (r[i].z<0.) {
            v[i].z *=-1; //- elastic walls
            psum += 2*m*fabs(v[i].z)/dt;  // contribution to pressure from "left" walls
        }
        else if (r[i].z>=L) {
            v[i].z*=-1;  //- elastic walls
            psum += 2*m*fabs(v[i].z)/dt;  // contribution to pressure from "right" walls
        }
    }
    
    
    // /* removed, uncomment to save atoms positions */
    // for (i=0; i<N; i++) {
    //     fprintf(fp,"%s",atype);
    //     fprintf(fp,"  %12.10e ",r[i].x);
    //     fprintf(fp,"  %12.10e ",r[i].y);
    //     fprintf(fp,"  %12.10e ",r[i].z);
    //     fprintf(fp,"\n");
    // }//*/
    // fprintf(fp,"\n \n");
    
    return psum/(6*L*L);
}

void initializeVelocities() {
    
    int i;
    
    for (i=0; i<N; i++) {
        v[i].x = gaussdist();
        v[i].y = gaussdist();
        v[i].z = gaussdist();
    }
    
    // Vcm = sum_i^N  m*v_i/  sum_i^N  M
    // Compute center-of-mas velocity according to the formula above
    Vect3d vCM = {0, 0, 0};
    
    for (i=0; i<N; i++) {
        vCM.x += m*v[i].x;
        vCM.y += m*v[i].y;
        vCM.z += m*v[i].z;
    }
    
    
    vCM.x /= (N*m);
    vCM.y /= (N*m);
    vCM.z /= (N*m);
    

    //  Subtract out the center-of-mass velocity from the
    //  velocity of each particle... effectively set the
    //  center of mass velocity to zero so that the system does
    //  not drift in space!
    // for (i=0; i<N; i++) {
    //     v[i].x -= vCM.x;
    //     v[i].y -= vCM.y;
    //     v[i].z -= vCM.z;
    // }

    
    //  Now we want to scale the average velocity of the system
    //  by a factor which is consistent with our initial temperature, Tinit
    double vSqdSum, lambda;
    vSqdSum=0.;
    Vect3d vel;
    for (i=0; i<N; i++) {
        v[i].x -= vCM.x;
        v[i].y -= vCM.y;
        v[i].z -= vCM.z;
        vel = v[i];
        vSqdSum += (vel.x*vel.x) + (vel.y*vel.y) + (vel.z*vel.z);
    }
    
    lambda = sqrt( 3*(N-1)*Tinit/vSqdSum);
    
    for (i=0; i<N; i++) {
        v[i].x *= lambda;
        v[i].y *= lambda;
        v[i].z *= lambda;
    }
}


//  Numerical recipes Gaussian distribution number generator
double gaussdist() {
    static bool available = false;
    static double gset;
    double fac, rsq, v1, v2;
    if (!available) {
        do {
            v1 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            v2 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            rsq = v1 * v1 + v2 * v2;
        } while (rsq >= 1.0 || rsq == 0.0);
        
        fac = sqrt(-2.0 * log(rsq) / rsq);
        gset = v1 * fac;
        available = true;
        
        return v2*fac;
    } else {
        
        available = false;
        return gset;
        
    }
}
